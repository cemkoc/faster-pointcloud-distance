#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <numeric>
//#include <pcl/gpu/containers/device_array.h>
#include <pcl/gpu/octree/octree.hpp>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "cloud_distance/distance.cuh"

#include <iostream>

#define NUM_THREADS 512

__global__ void compute_kernel(pcl::PointXYZ* cloud_a_ptr, pcl::PointXYZ* cloud_b_ptr,
    int asize, int bsize, double* mins) {
    int point_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (point_id >= asize) {
        return;
    }
    pcl::PointXYZ point_a = cloud_a_ptr[point_id];

    double min_sofar = std::numeric_limits<double>::max();
    for (int i = 0; i < bsize; ++i) {
        pcl::PointXYZ point_b = cloud_b_ptr[i];
        double dist = pow(point_a.x - point_b.x, 2.0) + pow(point_a.y - point_b.y, 2.0) + pow(point_a.z - point_b.z, 2.0);
        if (dist < min_sofar) {
            min_sofar = dist;
        }
    }
    mins[point_id] = min_sofar;
}

namespace distance {

double DistanceCuda::compute_distance(pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_a_ptr,
                                      pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_b_ptr) {
    pcl::PointCloud<pcl::PointXYZ> cloud_a = *cloud_a_ptr;
    pcl::PointCloud<pcl::PointXYZ> cloud_b = *cloud_b_ptr;

    // in order to upload, we need to essentially unalign the vector rip
    pcl::PointXYZ* local_a_ptr = new pcl::PointXYZ[cloud_a.size()];
    pcl::PointXYZ* local_b_ptr = new pcl::PointXYZ[cloud_b.size()];
    for (int i = 0; i < cloud_a.size(); ++i) {
        local_a_ptr[i] = cloud_a.points[i];
    }
    for (int i = 0; i < cloud_b.size(); ++i) {
        local_b_ptr[i] = cloud_b.points[i];
    }

    pcl::PointXYZ* cuda_a_ptr;
    pcl::PointXYZ* cuda_b_ptr;
    double* mins;
    hipMalloc((void**)&mins, max(cloud_a.size(), cloud_b.size()) * sizeof(double));
    hipMalloc((void**)&cuda_a_ptr, cloud_a.size() * sizeof(pcl::PointXYZ));
    hipMalloc((void**)&cuda_b_ptr, cloud_b.size() * sizeof(pcl::PointXYZ));

    hipMemcpy(cuda_a_ptr, local_a_ptr, cloud_a.size() * sizeof(pcl::PointXYZ), hipMemcpyHostToDevice);
    hipMemcpy(cuda_b_ptr, local_b_ptr, cloud_b.size() * sizeof(pcl::PointXYZ), hipMemcpyHostToDevice);

    int blks = (cloud_a.size() + NUM_THREADS - 1) / NUM_THREADS;
    compute_kernel<<<blks, NUM_THREADS>>>(cuda_a_ptr, cuda_b_ptr, cloud_a.size(), cloud_b.size(), mins);

    double sum_a = thrust::reduce(thrust::device, mins, mins + cloud_a.size(), 0.0);  // TODO THIS NEED TO BE DEVICE

    blks = (cloud_b.size() + NUM_THREADS - 1) / NUM_THREADS;
    compute_kernel<<<blks, NUM_THREADS>>>(cuda_b_ptr, cuda_a_ptr, cloud_b.size(), cloud_a.size(), mins);
    double sum_b = thrust::reduce(thrust::device, mins, mins + cloud_b.size(), 0.0);

    return (1.0 / cloud_a.size()) * sum_a + (1.0 / cloud_b.size()) * sum_b;
}

double DistanceCuda::compute_distance(pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_a_ptr,
                                      pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_b_ptr, int k) {

    printf("Running with K=%d\n", k);
    pcl::PointCloud<pcl::PointXYZ> cloud_a = *cloud_a_ptr;
    pcl::PointCloud<pcl::PointXYZ> cloud_b = *cloud_b_ptr;
    
    // can we reuse this for iteration? or do we need to upload 
    // the points separately
    pcl::gpu::Octree::PointCloud cloud_a_device;
    pcl::gpu::Octree::PointCloud cloud_b_device;
    cloud_a_device.upload(cloud_a.points);
    cloud_b_device.upload(cloud_b.points);

    pcl::gpu::Octree::Queries queries_a_device;
    pcl::gpu::Octree::Queries queries_b_device;

    // option 1
    queries_a_device.upload(cloud_a.points);
    queries_b_device.upload(cloud_b.points);
    //
    /* option 2
    std::vector<pcl::PointXYZ> points;
    points.resize(cloud_a.size());
    for (int i = 0; i < cloud_a.size(); ++i) {
        points[i] = cloud_a.points[i];
    }
    queries_a_device.upload(points);
    points.resize(cloud_b.size());
    for (int i = 0; i < cloud_b.size(); ++i) {
        points[i] = cloud_b.points[i];
    }
    queries_b_device.upload(points);
    */


    // reuse the same octree for both a and b
    pcl::gpu::Octree::Ptr octree_device (new pcl::gpu::Octree);
    octree_device->setCloud(cloud_b_device);
    octree_device->build();

    // can we reuse one of the _x_ind?
    pcl::gpu::NeighborIndices _a_ind(cloud_a.size(), 1);
    pcl::gpu::NeighborIndices _b_ind(cloud_b.size(), 1);
    pcl::gpu::Octree::ResultSqrDists a_res;
    pcl::gpu::Octree::ResultSqrDists b_res;

    // do the above need sizes???? TODO

    octree_device->nearestKSearchBatch(queries_a_device, 1, _a_ind, a_res);

    // can we sum on device instead?
    std::vector<float> downloaded(cloud_a.size());
    a_res.download(downloaded);
    double sum_a = accumulate(downloaded.begin(), downloaded.end(), 0.0);

    octree_device->setCloud(cloud_a_device);
    octree_device->build();
    octree_device->nearestKSearchBatch(queries_b_device, 1, _b_ind, b_res);

    // can we sum on device instead?
    downloaded.resize(cloud_b.size());
    b_res.download(downloaded);
    double sum_b = accumulate(downloaded.begin(), downloaded.end(), 0.0);

    return (1.0 / cloud_a.size()) * sum_a + (1.0 / cloud_b.size()) * sum_b;
}

/*
double DistanceCuda::compute_distance_radius(pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_a_ptr,
                                      pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_b_ptr) {

    printf("Running with K=%d\n", k);
    pcl::PointCloud<pcl::PointXYZ> cloud_a = *cloud_a_ptr;
    pcl::PointCloud<pcl::PointXYZ> cloud_b = *cloud_b_ptr;
    
    thrust::device_vector<float> mins(max(cloud_a.size(), cloud_b.size()));
    
    // can we reuse this for iteration? or do we need to upload 
    // the points separately
    pcl::gpu::Octree::PointCloud cloud_a_device;
    pcl::gpu::Octree::PointCloud cloud_b_device;
    cloud_a_device.upload(cloud_a.points);
    cloud_b_device.upload(cloud_b.points);

    pcl::gpu::Octree::Queries queries_a_device;
    pcl::gpu::Octree::Queries queries_b_device;

    // option 1
    queries_a_device.upload(cloud_a.points);
    queries_b_device.upload(cloud_b.points);
    //
    /* option 2
    std::vector<pcl::PointXYZ> points;
    points.resize(cloud_a.size());
    for (int i = 0; i < cloud_a.size(); ++i) {
        points[i] = cloud_a.points[i];
    }
    queries_a_device.upload(points);
    points.resize(cloud_b.size());
    for (int i = 0; i < cloud_b.size(); ++i) {
        points[i] = cloud_b.points[i];
    }
    queries_b_device.upload(points);
    //


    // reuse the same octree for both a and b
    pcl::gpu::Octree::Ptr octree_device (new pcl::gpu::Octree);
    octree_device->setCloud(cloud_b_device);
    octree_device->build();

    // can we reuse one of the _x_ind?
    pcl::gpu::NeighborIndices _a_ind(cloud_a.size(), 1);
    pcl::gpu::NeighborIndices _b_ind(cloud_b.size(), 1);

    // do the above need sizes???? TODO

    octree_device.nearestKSearchBatch(queries_a_device, 1, _a_ind, a_res);

    // can we sum on device instead?
    std::vector<float> downloaded(cloud_a.size());
    a_res.download(downloaded);
    double sum_a = accumulate(downloaded.begin(), downloaded.end(), 0.0);

    octree_device->setCloud(cloud_a_device);
    octree_device->build();
    octree_device.nearestKSearchBatch(queries_b_device, 1, _b_ind, b_res);

    // can we sum on device instead?
    downloaded.resize(cloud_b.size());
    b_res.download(downloaded);
    double sum_b = accumulate(downloaded.begin(), downloaded.end(), 0.0);

    return (1.0 / cloud_a.size()) * sum_a + (1.0 / cloud_b.size()) * sum_b;
}
*/
}
