#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <numeric>
//#include <pcl/gpu/containers/device_array.h>
#include <pcl/gpu/octree/octree.hpp>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "cloud_distance/distance.cuh"

#include <iostream>

#define NUM_THREADS 512

__global__ void naive_compute_kernel(pcl::PointXYZ* cloud_a_ptr, pcl::PointXYZ* cloud_b_ptr,
    int asize, int bsize, double* mins) {
    int point_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (point_id >= asize) {
        return;
    }
    pcl::PointXYZ point_a = cloud_a_ptr[point_id];

    double min_sofar = std::numeric_limits<double>::max();
    for (int i = 0; i < bsize; ++i) {
        pcl::PointXYZ point_b = cloud_b_ptr[i];
        double dist = pow(point_a.x - point_b.x, 2.0) + pow(point_a.y - point_b.y, 2.0) + pow(point_a.z - point_b.z, 2.0);
        if (dist < min_sofar) {
            min_sofar = dist;
        }
    }
    mins[point_id] = min_sofar;
}

__global__ void radius_compute_kernel(int size, int max_res, int* octree_indices, int* res_sizes,
                                      pcl::PointXYZ* octree_points, pcl::PointXYZ* search_points, double* mins) {
    int point_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (point_id >= size) {
        return;
    }
    if (!res_sizes[point_id]) {
        mins[point_id] = 0.0;
        return;
    }

    pcl::PointXYZ point_source = search_points[point_id];
    double min_sofar = std::numeric_limits<double>::max();
    for (int i = 0; i < res_sizes[point_id]; ++i) {
        pcl::PointXYZ radius_point = octree_points[octree_indices[point_id * max_res + i]];
        double dist = pow(point_source.x - radius_point.x, 2.0) + pow(point_source.y - radius_point.y, 2.0) + pow(point_source.z - radius_point.z, 2.0);
        if (dist < min_sofar) {
            min_sofar = dist;
        }
    }
    mins[point_id] = min_sofar;
}

namespace distance {

double DistanceCuda::compute_distance(pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_a_ptr,
                                      pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_b_ptr) {
    pcl::PointCloud<pcl::PointXYZ> cloud_a = *cloud_a_ptr;
    pcl::PointCloud<pcl::PointXYZ> cloud_b = *cloud_b_ptr;

    // in order to upload, we need to essentially unalign the vector rip
    pcl::PointXYZ* local_a_ptr = new pcl::PointXYZ[cloud_a.size()];
    pcl::PointXYZ* local_b_ptr = new pcl::PointXYZ[cloud_b.size()];
    for (int i = 0; i < cloud_a.size(); ++i) {
        local_a_ptr[i] = cloud_a.points[i];
    }
    for (int i = 0; i < cloud_b.size(); ++i) {
        local_b_ptr[i] = cloud_b.points[i];
    }

    pcl::PointXYZ* cuda_a_ptr;
    pcl::PointXYZ* cuda_b_ptr;
    double* mins;
    hipMalloc((void**)&mins, max(cloud_a.size(), cloud_b.size()) * sizeof(double));
    hipMalloc((void**)&cuda_a_ptr, cloud_a.size() * sizeof(pcl::PointXYZ));
    hipMalloc((void**)&cuda_b_ptr, cloud_b.size() * sizeof(pcl::PointXYZ));

    hipMemcpy(cuda_a_ptr, local_a_ptr, cloud_a.size() * sizeof(pcl::PointXYZ), hipMemcpyHostToDevice);
    hipMemcpy(cuda_b_ptr, local_b_ptr, cloud_b.size() * sizeof(pcl::PointXYZ), hipMemcpyHostToDevice);

    int blks = (cloud_a.size() + NUM_THREADS - 1) / NUM_THREADS;
    naive_compute_kernel<<<blks, NUM_THREADS>>>(cuda_a_ptr, cuda_b_ptr, cloud_a.size(), cloud_b.size(), mins);

    double sum_a = thrust::reduce(thrust::device, mins, mins + cloud_a.size(), 0.0);  // TODO THIS NEED TO BE DEVICE

    blks = (cloud_b.size() + NUM_THREADS - 1) / NUM_THREADS;
    naive_compute_kernel<<<blks, NUM_THREADS>>>(cuda_b_ptr, cuda_a_ptr, cloud_b.size(), cloud_a.size(), mins);
    double sum_b = thrust::reduce(thrust::device, mins, mins + cloud_b.size(), 0.0);

    return (1.0 / cloud_a.size()) * sum_a + (1.0 / cloud_b.size()) * sum_b;
}

double DistanceCuda::compute_distance(pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_a_ptr,
                                      pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_b_ptr, int k) {

    printf("Running with K=%d\n", k);
    pcl::PointCloud<pcl::PointXYZ> cloud_a = *cloud_a_ptr;
    pcl::PointCloud<pcl::PointXYZ> cloud_b = *cloud_b_ptr;
    
    // can we reuse this for iteration? or do we need to upload 
    // the points separately
    pcl::gpu::Octree::PointCloud cloud_a_device;
    pcl::gpu::Octree::PointCloud cloud_b_device;
    cloud_a_device.upload(cloud_a.points);
    cloud_b_device.upload(cloud_b.points);

    pcl::gpu::Octree::Ptr octree_device (new pcl::gpu::Octree);
    pcl::gpu::Octree::Queries queries_device;
    pcl::gpu::NeighborIndices _a_ind(cloud_a.size(), 1);
    pcl::gpu::NeighborIndices _b_ind(cloud_b.size(), 1);
    pcl::gpu::Octree::ResultSqrDists a_res;
    pcl::gpu::Octree::ResultSqrDists b_res;

    queries_device.upload(cloud_a.points);
    octree_device->setCloud(cloud_b_device);
    octree_device->build();

    octree_device->nearestKSearchBatch(queries_device, 1, _a_ind, a_res);

    queries_device.upload(cloud_b.points);

    // can we sum on device instead?
    std::vector<float> downloaded(cloud_a.size());
    a_res.download(downloaded);
    double sum_a = accumulate(downloaded.begin(), downloaded.end(), 0.0);

    octree_device->setCloud(cloud_a_device);
    octree_device->build();
    octree_device->nearestKSearchBatch(queries_device, 1, _b_ind, b_res);

    // can we sum on device instead?
    downloaded.resize(cloud_b.size());
    b_res.download(downloaded);
    double sum_b = accumulate(downloaded.begin(), downloaded.end(), 0.0);

    return (1.0 / cloud_a.size()) * sum_a + (1.0 / cloud_b.size()) * sum_b;
}

double DistanceCuda::compute_distance_radius(pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_a_ptr,
                                      pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_b_ptr) {

    float rad = 3;
    int max_res = 1024;
    printf("Running with Radius=%f\n", rad);
    pcl::PointCloud<pcl::PointXYZ> cloud_a = *cloud_a_ptr;
    pcl::PointCloud<pcl::PointXYZ> cloud_b = *cloud_b_ptr;
    double* mins;
    hipMalloc((void**)&mins, max(cloud_a.size(), cloud_b.size()) * sizeof(double));

    // can we reuse this for iteration? or do we need to upload 
    // the points separately
    pcl::gpu::Octree::PointCloud cloud_a_device;
    pcl::gpu::Octree::PointCloud cloud_b_device;
    cloud_a_device.upload(cloud_a.points);
    cloud_b_device.upload(cloud_b.points);

    pcl::gpu::Octree::Queries queries_device;
    pcl::gpu::Octree::Ptr octree_device (new pcl::gpu::Octree);
    pcl::gpu::NeighborIndices a_ind(cloud_a.size(), max_res);
    pcl::gpu::NeighborIndices b_ind(cloud_b.size(), max_res);

    queries_device.upload(cloud_a.points);
    octree_device->setCloud(cloud_b_device);
    octree_device->build();

    octree_device->radiusSearch(queries_device, rad, max_res, a_ind);
    int blks = (cloud_a.size() + NUM_THREADS - 1) / NUM_THREADS;
    radius_compute_kernel<<<blks, NUM_THREADS>>>(cloud_a.size(), max_res, a_ind.data.ptr(), a_ind.sizes.ptr(),
                                                 cloud_b_device.ptr(), cloud_a_device.ptr(), mins);

    double sum_a = thrust::reduce(thrust::device, mins, mins + cloud_a.size(), 0.0);

    queries_device.upload(cloud_b.points);
    octree_device->setCloud(cloud_a_device);
    octree_device->build();

    octree_device->radiusSearch(queries_device, rad, max_res, b_ind);
    blks = (cloud_b.size() + NUM_THREADS - 1) / NUM_THREADS;
    radius_compute_kernel<<<blks, NUM_THREADS>>>(cloud_b.size(), max_res, b_ind.data.ptr(), b_ind.sizes.ptr(),
                                                 cloud_a_device.ptr(), cloud_b_device.ptr(), mins);
                                                 
    double sum_b = thrust::reduce(thrust::device, mins, mins + cloud_b.size(), 0.0);

    return (1.0 / cloud_a.size()) * sum_a + (1.0 / cloud_b.size()) * sum_b;
}

}
