#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>
#include <numeric>
#include <pcl/gpu/containers/device_array.h>
#include <pcl/gpu/octree/octree.hpp>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include <pcl/cloud_distance/distance.cuh>

#include <iostream>

#define NUM_THREADS 512

__global__ void compute_kernel(pcl::PointXYZ* cloud_a_ptr, pcl::PointXYZ* cloud_b_ptr,
    int asize, int bsize, double* mins) {
    int point_id = threadIdx.x + blockIdx.x * blockDim.x;
    if (point_id >= asize) {
        return;
    }
    pcl::PointXYZ point_a = cloud_a_ptr[point_id];

    double min_sofar = std::numeric_limits<double>::max();
    for (int i = 0; i < bsize; ++i) {
        pcl::PointXYZ point_b = cloud_b_ptr[i];
        double dist = pow(point_a.x - point_b.x, 2.0) + pow(point_a.y - point_b.y, 2.0) + pow(point_a.z - point_b.z, 2.0);
        if (dist < min_sofar) {
            min_sofar = dist;
        }
    }
    if (point_id % 10000 == 0) {
        printf("reached after\n");
    }
    mins[point_id] = min_sofar;
}

namespace distance {

double DistanceCuda::compute_distance(pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_a_ptr,
                                      pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_b_ptr) {
    pcl::PointCloud<pcl::PointXYZ> cloud_a = *cloud_a_ptr;
    pcl::PointCloud<pcl::PointXYZ> cloud_b = *cloud_b_ptr;

    pcl::PointXYZ* cuda_a_ptr;
    pcl::PointXYZ* cuda_b_ptr;
    double* mins;
    hipMalloc((void**)&mins, max(cloud_a.points.size(), cloud_b.points.size()) * sizeof(double));
    hipMalloc((void**)&cuda_a_ptr, cloud_a.points.size() * sizeof(pcl::PointXYZ));
    hipMalloc((void**)&cuda_b_ptr, cloud_b.points.size() * sizeof(pcl::PointXYZ));
    std::cout << "blah" << std::endl;
    hipMemcpy(cuda_a_ptr, &(cloud_a.points), cloud_a.points.size() * sizeof(pcl::PointXYZ), hipMemcpyHostToDevice);

    std::cout << "a alloced" << std::endl;
    hipMemcpy(cuda_b_ptr, &(cloud_b.points), cloud_b.points.size() * sizeof(pcl::PointXYZ), hipMemcpyHostToDevice);
    std::cout << "finished allocs" << std::endl;

    int blks = (cloud_a.points.size() + NUM_THREADS - 1) / NUM_THREADS;
    compute_kernel<<<blks, NUM_THREADS>>>(cuda_a_ptr, cuda_b_ptr, cloud_a.points.size(), cloud_b.points.size(), mins);
    hipDeviceSynchronize();
    std::cout << "first calc done" << std::endl;
    double sum_a = thrust::reduce(thrust::device, mins, mins + cloud_a.points.size(), 0.0);  // TODO THIS NEED TO BE DEVICE

    std::cout << "first red done" << std::endl;

    blks = (cloud_b.points.size() + NUM_THREADS - 1) / NUM_THREADS;
    compute_kernel<<<blks, NUM_THREADS>>>(cuda_b_ptr, cuda_a_ptr, cloud_b.points.size(), cloud_a.points.size(), mins);
    std::cout << "second calc done" << std::endl;
    double sum_b = thrust::reduce(thrust::device, mins, mins + cloud_b.points.size(), 0.0);

    return (1.0 / cloud_a.size()) * sum_a + (1.0 / cloud_b.size()) * sum_b;
}

// double DistanceCuda::compute_distance(pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_a_ptr,
//                                       pcl::PointCloud<pcl::PointXYZ>::ConstPtr cloud_b_ptr, int k) {

//     printf("Running with K=%d\n", k);
//     pcl::PointCloud<pcl::PointXYZ> cloud_a = *cloud_a_ptr;
//     pcl::PointCloud<pcl::PointXYZ> cloud_b = *cloud_b_ptr;
    
//     thrust::device_vector<float> mins(max(cloud_a.points.size(), cloud_b.points.size()));
//     // can we reuse this for iteration? or do we need to upload 
//     // the points separately
//     pcl::gpu::Octree::PointCloud cloud_a_device;
//     pcl::gpu::Octree::PointCloud cloud_b_device;
//     cloud_a_device.upload(cloud_a.points);
//     cloud_b_device.upload(cloud_b.points);

//     pcl::gpu::Octree::Queries queries_a_device = cloud_a_device;
//     pcl::gpu::Octree::Queries queries_b_device = cloud_b_device;

//     // reuse the same octree for both a and b
//     pcl::gpu::Octree::Ptr octree_device (new pcl::gpu::Octree);
//     octree_device->setCloud(cloud_b_device);
//     octree_device->build();

//     // can we reuse one of the _x_ind?
//     pcl::gpu::NeighborIndices _a_ind(cloud_a.size(), 1);
//     pcl::gpu::NeighborIndices _b_ind(cloud_b.size(), 1);
//     pcl::gpu::Octree::ResultSqrDists a_res;
//     pcl::gpu::Octree::ResultSqrDists b_res;

//     octree_device.nearestKSearchBatch(queries_a_device, 1, _a_ind, a_res);

//     // can we sum on device instead?
//     std::vector<float> downloaded(cloud_a.size());
//     a_res.download(downloaded);
//     double sum_a = accumulate(downloaded.begin(), downloaded.end(), 0.0);

//     octree_device->setCloud(cloud_a_device);
//     octree_device->build();
//     octree_device.nearestKSearchBatch(queries_b_device, 1, _b_ind, b_res);

//     // can we sum on device instead?
//     downloaded.resize(cloud_b.size());
//     a_res.download(downloaded);
//     double sum_b = accumulate(downloaded.begin(), downloaded.end(), 0.0);

//     /* or should we do this method/ is it possible?
//     compute_kernel(cloud_a_device, octree_device, mins);
//     double sum_a = thrust::reduce(mins.begin(), mins.begin() + cloud_a.points.size(), thrust::plus<float>());

//     octree_device->setCloud(cloud_a_device);
//     octree_device->build();

//     compute_kernel(cloud_b_device, octree_device, mins);
//     double sum_b = thrust::reduce(mins.begin(), mins.begin() + cloud_b.points.size(), thrust::plus<float>());
//     */
//     return (1.0 / cloud_a.size()) * sum_a + (1.0 / cloud_b.size()) * sum_b;
// }

// // __global__ void compute_kernel(pcl::gpu::Octree::PointCloud cloud_a, pcl::gpu::Octree::Ptr octree_ptr,
// //                                thrust::device_vector& mins) {
// //     int point_id = threadIdx.x + blockIdx.x * blockDim.x;
// //     if (point_id > asize) {
// //         return;
// //     }
// //     pcl::
// // }
}